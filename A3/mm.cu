
#include <hip/hip_runtime.h>
#include <math.h>

#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>

#ifndef N
#define N 4
#endif

#ifndef TEST
#define TEST 0
#endif

#ifndef TPB
#define TPB 1024
#endif

using namespace std;

void print(float* M);

float randomFloat() {
    static mt19937 generator(42);
    static uniform_real_distribution<double> dist(-2, 2);
    return dist(generator);
}

__global__ void multiply(float* A, float* B, float* C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N * N) return;
    int i = idx / N, j = idx % N;
    for (int k = 0; k < N; k++) C[i * N + j] += A[i * N + k] * B[k * N + j];
    return;
}

int main() {
    typedef std::chrono::high_resolution_clock Time;
    typedef std::chrono::duration<float> fsec;

    static float A[N * N], B[N * N], C[N * N] = {0};
    int i, j;
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++) {
            A[i * N + j] = randomFloat();
            B[i * N + j] = randomFloat();
        }

    float *A_dev, *B_dev, *C_dev;
    int mat_size = N * N * sizeof(float);
    int num_blocks = ceil(float(N * N) / TPB);

    hipMalloc((void**)&A_dev, mat_size);
    hipMalloc((void**)&B_dev, mat_size);
    hipMalloc((void**)&C_dev, mat_size);
    
    auto start = Time::now();
    hipMemcpy(A_dev, A, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B, mat_size, hipMemcpyHostToDevice);

    multiply<<<num_blocks, TPB>>>(A_dev, B_dev, C_dev);
    hipMemcpy(C, C_dev, mat_size, hipMemcpyDeviceToHost);

    auto stop = Time::now();
    fsec timer1 = stop - start;
#if TEST
    print(C);
#endif
    // cout << C[N*N-1] << endl;
    cout << timer1.count() * 1000 << " ms" << endl;
}

void print(float* M) {
    cout << endl;
    cout.setf(ios::fixed);
    for (int i = 0; i < N; i++) {
        cout << '\t';
        for (int j = 0; j < N; j++)
            cout << setw(10) << setprecision(2) << M[i * N + j] << " ";
        cout << endl;
    }
    cout << endl;
}
